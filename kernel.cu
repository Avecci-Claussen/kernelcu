#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <stdbool.h>

#include "kernel.cuh"
#include "sha256.cuh"
#include "../requests.cuh"
#include "../crypto/hex.cuh"

#define TOTAL_SIZE 108
#define MAX_SHARES 16

#define MIN(x, y) (((x) < (y)) ? (x) : (y))
#define CLEAR() printf("\033[H\033[J")

__device__ __constant__ char share_chunk_c[64];
__device__ __constant__ size_t share_difficulty_c;

__device__ __constant__ char shared_digits[] = "0123456789abcdef";

__device__ __forceinline__ void sha256_to_hex(unsigned char *hash, char *hex) {
#pragma unroll
    for (int i = 0; i < 16; ++i) {
        char lo_nibble = shared_digits[hash[i] & 0x0F];
        char hi_nibble = shared_digits[(hash[i] & 0xF0) >> 4];
        *hex++ = hi_nibble;
        *hex++ = lo_nibble;
    }
    *hex = '\0';
}

__device__ __forceinline__ bool is_valid(const char* str) {
    int mask = 0;
#pragma unroll
    for (int i = 0; i < share_difficulty_c; ++i) {
        mask |= (str[i] ^ share_chunk_c[i]);
    }
    return mask == 0;
}

__global__ void miner(unsigned char **out, bool *stop, unsigned char *prefix, int *share_id) {
    const uint32_t tid = threadIdx.x;
    __shared__ SHA256_CTX shared_prefix_ctx;

    if (tid == 0) {
        sha256_init_dev(&shared_prefix_ctx);
        sha256_update_dev(&shared_prefix_ctx, prefix, sizeof(unsigned char) * (TOTAL_SIZE-4));
    }
    __syncthreads();

    unsigned char _hex[TOTAL_SIZE];
    memcpy(_hex, prefix, sizeof(unsigned char) * (TOTAL_SIZE-4));

    SHA256_CTX ctx;
    unsigned char hash[32];
    char hash_hex[64];

    for (uint32_t index = blockIdx.x * blockDim.x + tid; !(*stop); index += blockDim.x * gridDim.x) {
        _hex[TOTAL_SIZE-1] = index;
        _hex[TOTAL_SIZE-2] = index >> 8;
        _hex[TOTAL_SIZE-3] = index >> 16;
        _hex[TOTAL_SIZE-4] = index >> 24;

        memcpy(&ctx, &shared_prefix_ctx, sizeof(SHA256_CTX));

        sha256_update_dev(&ctx, _hex + (TOTAL_SIZE-4), sizeof(unsigned char) * 4);
        sha256_final_dev(&ctx, hash);
        sha256_to_hex(hash, hash_hex);

        if (is_valid(hash_hex)) {
            int id = atomicAdd(share_id, 1);
            memcpy(out[id], _hex, sizeof(unsigned char) * TOTAL_SIZE);

            if (id >= MAX_SHARES-2) {
                *stop = true;
            }
        }

        if (index >= 0xFFFFFFFF) {
            *stop = true;
        }
    }
}

void start(GpuSettings *settings, ManagerData *managerData) {
    auto res = hipSetDevice(settings->deviceId);
    if (res != hipSuccess) {
        printf("Error setting device: %s\n", hipGetErrorString(res));
        return;
    }

    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, settings->deviceId);

    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(dev_k), host_k, sizeof(host_k), 0, hipMemcpyHostToDevice));

    int zero = 0;
    bool *stop_g;
    hipMallocManaged(&stop_g, sizeof(bool));
    hipMemcpy(stop_g, &zero, sizeof(bool), hipMemcpyHostToDevice);

    int *share_id;
    hipMallocManaged(&share_id, sizeof(int));
    hipMemcpy(share_id, &zero, sizeof(int), hipMemcpyHostToDevice);

    unsigned char *prefix_g;
    hipMallocManaged(&prefix_g, sizeof(unsigned char) * (TOTAL_SIZE-4));

    unsigned char **out_g;
    hipMallocManaged(&out_g, sizeof(unsigned char*) * MAX_SHARES);
    for (int i = 0; i < MAX_SHARES; ++i) {
        hipMallocManaged(&out_g[i], sizeof(unsigned char) * TOTAL_SIZE);
        hipMemset(out_g[i], 0, sizeof(unsigned char) * TOTAL_SIZE);
    }

    hipMemcpyToSymbol(HIP_SYMBOL(share_chunk_c), managerData->shareChunk, sizeof(char) * 64);
    hipMemcpyToSymbol(HIP_SYMBOL(share_difficulty_c), &settings->shareDifficulty, sizeof(size_t));

    size_t num_threads = min(static_cast<size_t>(256), static_cast<size_t>(deviceProp.maxThreadsPerBlock));
    size_t num_blocks = min(static_cast<size_t>(1012), (deviceProp.multiProcessorCount * deviceProp.maxThreadsPerMultiProcessor) / num_threads);

    hipError_t err;
    hipEvent_t start, end;
    uint loops_count = 0;

    err = hipEventCreate(&start);
    err = hipEventCreate(&end);

    while (!(*managerData->stop)) {
        float elapsed_ms = 0.0f;

        err = hipEventRecord(start, 0);

        time_t now = time(NULL);
        hipMemcpy(prefix_g, managerData->prefix, sizeof(unsigned char) * (TOTAL_SIZE-4), hipMemcpyHostToDevice);

        prefix_g[98] = now & 0xFF;
        prefix_g[99] = (now >> 8) & 0xFF;
        prefix_g[100] = (now >> 16) & 0xFF;
        prefix_g[101] = (now >> 24) & 0xFF;

        miner<<<num_blocks, num_threads>>> (out_g, stop_g, prefix_g, share_id);
        checkCudaErrors(hipDeviceSynchronize());

        err = hipEventRecord(end, 0);
        err = hipEventSynchronize(end);
        err = hipEventElapsedTime(&elapsed_ms, start, end);

        if (!settings->silent) {
            float hashrate = 4294967296.0 / (elapsed_ms / 1000.0) / 1000000000.0;
            CLEAR();
            printf("Denaro GPU Miner\n\n");
            printf("Device: %s\n", deviceProp.name);
            printf("Threads: %d\n", num_threads);
            printf("Blocks: %d\n\n", num_blocks);
            printf("Node: %s\n", settings->nodeUrl);
            printf("Pool: %s\n\n", settings->poolUrl);
            printf("Accepted shares: %d\n\n", managerData->shares);
            printf("Hashrate: %.2f GH/s\n", hashrate);
        }

        if (*share_id > 0) {
            Share resp;
            unsigned char *out;
            hipMallocManaged(&out, sizeof(unsigned char) * TOTAL_SIZE);
            for (int i = 0; i < MIN(*share_id, MAX_SHARES); ++i) {
                hipMemcpy(out, out_g[i], sizeof(unsigned char) * TOTAL_SIZE, hipMemcpyDeviceToHost);
                if (out[0] == 2) {
                    resp = share(settings->poolUrl, bin2hex(out, TOTAL_SIZE), managerData->miningInfo.result.pending_transactions_hashes, managerData->miningInfo.result.pending_transactions_count, managerData->miningInfo.result.last_block.id + 1);
                    if (resp.ok) {
                        if (settings->verbose) {
                            printf("Share accepted: %s\n", bin2hex(out, TOTAL_SIZE));
                        }
                        managerData->shares++;
                    } else {
                        if (settings->verbose) {
                            printf("Share not accepted: %s\n", resp.error);
                        }
                        *managerData->stop = true;
                    }
                }
                hipMemset(out_g[i], 0, sizeof(unsigned char) * TOTAL_SIZE);
            }
            *share_id = 0;
        }

        *stop_g = false;
        loops_count++;
    }

    for (int i = 0; i < MAX_SHARES; ++i) {
        hipFree(out_g[i]);
    }
    hipFree(out_g);
    hipFree(stop_g);
    hipFree(share_id);
    hipFree(prefix_g);
    hipEventDestroy(start);
    hipEventDestroy(end);
    hipDeviceReset();
}
